#include "model.h"
#include "../cuda_err.h"

void Model::loadToGPU(Model *gpu_model)
{
    Model cpu_copy = *this;
    cpu_copy.triangles = nullptr;
    gpu_model = nullptr;
    //Выделить память под массив треугольников на GPU, скопировать
    gpuErrchk(hipMalloc(&(cpu_copy.triangles), num_of_triangles * sizeof(Triangle)));
    gpuErrchk(hipMemcpy(cpu_copy.triangles, this->triangles,
                         num_of_triangles * sizeof(Triangle), hipMemcpyHostToDevice));
    //Выделить память под структуру на GPU и отправить туда ее копию
    gpuErrchk(hipMalloc(&gpu_model, sizeof(Model)));
    gpuErrchk(hipMemcpy(gpu_model, &cpu_copy, sizeof(Model), hipMemcpyHostToDevice));
}

void Model::destroyGPU(Model *gpu_model)
{
    if (gpu_model != nullptr) {
        Model cpu_copy;
        gpuErrchk(hipMemcpy(&cpu_copy, gpu_model, sizeof(Model), hipMemcpyDeviceToHost));
        if (cpu_copy.triangles != nullptr) {
            gpuErrchk(hipFree(cpu_copy.triangles));
        }
        gpuErrchk(hipFree(gpu_model));
    }
}
