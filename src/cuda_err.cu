#include <stdio.h>
#include "cuda_err.h"

bool gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code == hipSuccess) return false;
   else {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
      return true;
   }
}
